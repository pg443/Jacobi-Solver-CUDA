#include "hip/hip_runtime.h"
#include "jacobi_iteration.h"

/* FIXME: Write the device kernels to solve the Jacobi iterations */

__global__ void jacobi_iteration_kernel_naive(matrix_t Ad, matrix_t Bd, matrix_t Xd, double* globalSSD)
{
    __shared__ double sum_per_thread[THREAD_BLOCK_SIZE];    /* Shared memory for thread block */
    int i =  blockIdx.x * blockDim.x + threadIdx.x;
    if (i > Xd.num_rows-1){
        sum_per_thread[threadIdx.x] = 0;
        return;
    }

    int num_cols = Ad.num_columns;
    float new_x;
    double sum = -Ad.elements[i * num_cols + i] * Xd.elements[i];
    for (int j = 0; j < num_cols; j++){
        sum += Ad.elements[i * num_cols + j] * Xd.elements[j];
    }
    new_x = (Bd.elements[i] - sum)/Ad.elements[i * num_cols + i];
    sum_per_thread[threadIdx.x] = (new_x - Xd.elements[i]) * (new_x - Xd.elements[i]);
    __syncthreads();

    int j = blockDim.x/2;
    while (j != 0) {
        if (threadIdx.x < j)
            sum_per_thread[threadIdx.x] += sum_per_thread[threadIdx.x + j];
        __syncthreads();
        j /= 2;
    }

    Xd.elements[i] = new_x;
    /* Check for convergence and update the unknowns. */
    if (threadIdx.x == 0){
        atomicAdd(globalSSD, sum_per_thread[0]);
    }

    return;
}

__global__ void jacobi_iteration_kernel_optimized(matrix_t Ad, matrix_t Bd, matrix_t Xd, double* globalSSD)
{
    __shared__ double sum_per_thread[THREAD_BLOCK_SIZE];    /* Shared memory for thread block */
    int i =  blockIdx.x * blockDim.x + threadIdx.x;
    if (i > Xd.num_rows-1){
        sum_per_thread[threadIdx.x] = 0;
        return;
    }

    int num_cols = Ad.num_columns;
    float new_x;
    double sum = -Ad.elements[i * num_cols + i] * Xd.elements[i];
    for (int j = 0; j < num_cols; j++){
        sum += Ad.elements[j * num_cols + i] * Xd.elements[j];
    }
    new_x = (Bd.elements[i] - sum)/Ad.elements[i * num_cols + i];
    sum_per_thread[threadIdx.x] = (new_x - Xd.elements[i]) * (new_x - Xd.elements[i]);
    __syncthreads();

    int j = blockDim.x/2;
    while (j != 0) {
        if (threadIdx.x < j)
            sum_per_thread[threadIdx.x] += sum_per_thread[threadIdx.x + j];
        __syncthreads();
        j /= 2;
    }

    Xd.elements[i] = new_x;
    /* Check for convergence and update the unknowns. */
    if (threadIdx.x == 0){
        atomicAdd(globalSSD, sum_per_thread[0]);
    }
    return;
}

